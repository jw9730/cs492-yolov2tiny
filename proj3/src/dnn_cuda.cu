#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 512

#define INDEX_ROW_MAJOR_3(i, j, k, I, J, K) (k + K * (j + J * (i)))
#define INDEX_ROW_MAJOR_4(i, j, k, l, I, J, K, L) (l + L * (k + K * (j + J * (i))))

#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))
static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString( err ), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void conv_ws(float *I, float *K, float *R, int iw, int ih, int ow, int oh, int kw, int kh, int sw, int sh, int ic, int oc){
    int BLOCKS_PER_PIXEL = ceil(float(oc)/float(THREADS_PER_BLOCK));
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int cid = blockIdx.x % BLOCKS_PER_PIXEL;
    int pid = blockIdx.x / BLOCKS_PER_PIXEL;
    // compute block index in output channel dimension
    int ofs = cid * THREADS_PER_BLOCK;
    int n_tid = (oc - ofs < THREADS_PER_BLOCK)? (oc - ofs) : THREADS_PER_BLOCK;
    // compute output pixel of the block
    int h = pid % oh;
    int w = pid / oh;
    if (tid >= n_tid) return;
    // declare on-chip shared memory
    extern __shared__ float M[];
    // read input data once per block (shared across threads)
    if(threadIdx.x == 0){
        for (int i=0; i<kw; i++){
            for (int j=0; j<kh; j++){
                for (int k=0; k<ic; k++){
                    M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] = I[INDEX_ROW_MAJOR_3(w*sw+i,h*sh+j,k, iw,ih,ic)];
                }
            }
        }
    }
    // wait until data is ready
    __syncthreads();
    // apply convolution
    float *o = R + INDEX_ROW_MAJOR_3(w,h,ofs+tid, ow,oh,oc);
    for (int i=0; i<kw; i++){
        for (int j=0; j<kh; j++){
            for (int k=0; k<ic; k++){
                atomicAdd(o, M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] * K[INDEX_ROW_MAJOR_4(i,j,k,ofs+tid, kw,kh,ic,oc)]);
            }
        }
    }
}

extern "C"
void conv2d(float * I, float * K, float * R, int iw, int ih, int ow, int oh, int kw, int kh, int sw, int sh, int ic, int oc) {
    float *dev_I, *dev_K, *dev_R;
    // I: (iw * ih * ic), row major ordered
    // K: (kw * kh * ic * oc), row major ordered
    // R: (ow * oh * oc), row major ordered
    // todo: 2d convolution between I and K
    // loop over outer dimensions, and compute dot product in chunks of size 512
    // kernel function: convolution for a single sliding window
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_I, iw * ih * ic * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_K, kw * kh * ic * oc * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_R, ow * oh * oc * sizeof(float) ) );
    // copy the arrays to the GPU
    HANDLE_ERROR( hipMemcpy( dev_I, I, iw * ih * ic * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_K, K, kw * kh * ic * oc * sizeof(float), hipMemcpyHostToDevice ) );
    // how to organiza blocks?
    // maximizing data reuse, spatial locality
    // thread over output channels (input stationary)
    int BLOCKS_PER_PIXEL = ceil(float(oc)/float(THREADS_PER_BLOCK));
    int BLOCKS = ow * oh * BLOCKS_PER_PIXEL;
    int BLOCK_MEMSIZE = kw * kh * ic * sizeof(float);
    conv_ws<<<BLOCKS,THREADS_PER_BLOCK,BLOCK_MEMSIZE>>>(dev_I, dev_K, dev_R, iw, ih, ow, oh, kw, kh, sw, sh, ic, oc);
    // copy the array back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( R, dev_R, ow * oh * oc * sizeof(float), hipMemcpyDeviceToHost ) );
    // cleanup
    hipFree(dev_I); hipFree(dev_K); hipFree(dev_R);
}