#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 512
#define INDEX_ROW_MAJOR_3(i, j, k, I, J, K) (k + K * (j + J * (i)))
#define INDEX_ROW_MAJOR_4(i, j, k, l, I, J, K, L) (l + L * (k + K * (j + J * (i))))

#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))
static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString( err ), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void conv_is(float *I, float *K, float *R, int iw, int ih, int ow, int oh, int kw, int kh, int sw, int sh, int ic, int oc){
    // input stationary
    int BLOCKS_PER_PIXEL = ceil(float(oc)/float(THREADS_PER_BLOCK));
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int cid = bid % BLOCKS_PER_PIXEL; // channel block index (within pixel)
    int pid = bid / BLOCKS_PER_PIXEL; // pixel index
    // compute block index in output channel dimension
    int ofs = cid * THREADS_PER_BLOCK;
    int n_tid = (oc - ofs < THREADS_PER_BLOCK)? (oc - ofs) : THREADS_PER_BLOCK;
    // compute output pixel of the block
    int h = pid % oh;
    int w = pid / oh;
    // declare on-chip shared memory
    extern __shared__ float M[];
    // read input data once per block (shared across threads)
    // this process could serve as bottleneck, load distribution is critical
    // distribute indices across threads
    int full_idx = kw * kh * ic;
    int load_per_thread = ceil(float(full_idx)/float(THREADS_PER_BLOCK));
    int lower = load_per_thread * tid;
    int upper = load_per_thread * (tid + 1);
    if (lower < full_idx) {
        upper = (upper < full_idx)? upper : full_idx;
        for (int idx=lower; idx<upper; idx++){
            int k = idx%ic;
            int j = idx/ic%kh;
            int i = idx/ic/kh;
            M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] = I[INDEX_ROW_MAJOR_3(w*sw+i,h*sh+j,k, iw,ih,ic)];
        }
    }
    /*
    if(tid == 0){
        for (int i=0; i<kw; i++){
            for (int j=0; j<kh; j++){
                for (int k=0; k<ic; k++){
                    M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] = I[INDEX_ROW_MAJOR_3(w*sw+i,h*sh+j,k, iw,ih,ic)];
                }
            }
        }
    }
    */
    // wait until data is ready
    __syncthreads();
    // handle boundary
    if (tid >= n_tid) return;
    // apply convolution
    float *o = R + INDEX_ROW_MAJOR_3(w,h,ofs+tid, ow,oh,oc);
    for (int i=0; i<kw; i++){
        for (int j=0; j<kh; j++){
            for (int k=0; k<ic; k++){
                atomicAdd(o, M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] * K[INDEX_ROW_MAJOR_4(i,j,k,ofs+tid, kw,kh,ic,oc)]);
            }
        }
    }
}
__global__ void conv_ws(float *I, float *K, float *R, int iw, int ih, int ow, int oh, int kw, int kh, int sw, int sh, int ic, int oc){
    // weight stationary
    int BLOCKS_PER_CHANNEL = ceil(float(ow * oh)/float(THREADS_PER_BLOCK));
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int pid = bid % BLOCKS_PER_CHANNEL; // pixel block index (within channel)
    int cid = bid / BLOCKS_PER_CHANNEL; // output channel index
    // compute block index in output pixel dimension
    int ofs = pid * THREADS_PER_BLOCK;
    int n_tid = (ow * oh - ofs < THREADS_PER_BLOCK)? (ow * oh - ofs) : THREADS_PER_BLOCK;
    // declare on-chip shared memory
    extern __shared__ float M[];
    // read kernel weight once per block (shared across threads)
    // this process could serve as bottleneck, load distribution is critical
    // distribute indices across threads
    if(tid == 0){
        for (int i=0; i<kw; i++){
            for (int j=0; j<kh; j++){
                for (int k=0; k<ic; k++){
                    M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] = K[INDEX_ROW_MAJOR_4(i,j,k,cid, iw,ih,ic,oc)];
                }
            }
        }
    }
    // wait until data is ready
    __syncthreads();
    // handle boundary
    if (tid >= n_tid) return;
    // apply convolution
    // retrieve output pixel
    int pos = ofs + tid;
    int w = pos/oc/oh;
    int h = pos/oc%oh;
    printf("[%d, %d, %d], bid %d, tid %d/%d, BLOCKS_PER_CHANNEL %d\n", w, h, cid, tid, bid, n_tid-1, BLOCKS_PER_CHANNEL);
    float *o = R + INDEX_ROW_MAJOR_3(w,h,cid, ow,oh,oc);
    for (int i=0; i<kw; i++){
        for (int j=0; j<kh; j++){
            for (int k=0; k<ic; k++){
                atomicAdd(o, I[INDEX_ROW_MAJOR_3(w*sw+i,h*sh+j,k, kw,kh,ic)] * M[INDEX_ROW_MAJOR_4(i,j,k,cid, kw,kh,ic,oc)]);
            }
        }
    }
}
extern "C"
void conv2d(float * I, float * K, float * R, int iw, int ih, int ow, int oh, int kw, int kh, int sw, int sh, int ic, int oc) {
    float *dev_I, *dev_K, *dev_R;
    // I: (iw * ih * ic), row major ordered
    // K: (kw * kh * ic * oc), row major ordered
    // R: (ow * oh * oc), row major ordered
    // todo: 2d convolution between I and K
    // loop over outer dimensions, and compute dot product in chunks of size 512
    // kernel function: convolution for a single sliding window
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_I, iw * ih * ic * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_K, kw * kh * ic * oc * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_R, ow * oh * oc * sizeof(float) ) );
    // copy the arrays to the GPU
    HANDLE_ERROR( hipMemcpy( dev_I, I, iw * ih * ic * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_K, K, kw * kh * ic * oc * sizeof(float), hipMemcpyHostToDevice ) );
    // how to organize blocks?
    // maximizing data reuse and parallelism within a block
    if (oc > THREADS_PER_BLOCK){
        // input stationary
        // within a block, hold input and thread over output channels
        int BLOCKS_PER_PIXEL = ceil(float(oc)/float(THREADS_PER_BLOCK));
        int BLOCKS = ow * oh * BLOCKS_PER_PIXEL;
        int BLOCK_MEMSIZE = kw * kh * ic * sizeof(float);
        conv_is<<<BLOCKS,THREADS_PER_BLOCK,BLOCK_MEMSIZE>>>(dev_I, dev_K, dev_R, iw, ih, ow, oh, kw, kh, sw, sh, ic, oc);
    }else{
        // weight stationary
        // within a block, hold kernel and thread over output pixels
        int BLOCKS_PER_CHANNEL = ceil(float(ow * oh)/float(THREADS_PER_BLOCK));
        int BLOCKS = oc * BLOCKS_PER_CHANNEL;
        int BLOCK_MEMSIZE = kw * kh * ic * sizeof(float);
        conv_ws<<<BLOCKS,THREADS_PER_BLOCK,BLOCK_MEMSIZE>>>(dev_I, dev_K, dev_R, iw, ih, ow, oh, kw, kh, sw, sh, ic, oc);
    }
    // copy the array back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( R, dev_R, ow * oh * oc * sizeof(float), hipMemcpyDeviceToHost ) );
    // cleanup
    hipFree(dev_I); hipFree(dev_K); hipFree(dev_R);
}