#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 512
#define INDEX_ROW_MAJOR_2(i, j, I, J) ((j) + (J) * (i))
#define INDEX_ROW_MAJOR_3(i, j, k, I, J, K) ((k) + (K) * ((j) + (J) * (i)))
#define INDEX_ROW_MAJOR_4(i, j, k, l, I, J, K, L) ((l) + (L) * ((k) + (K) * ((j) + (J) * (i))))

#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))
static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString( err ), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void conv_is(float *I, float *K, float *R, int iw, int ih, int ow, int oh, int kw, int kh, int sw, int sh, int ic, int oc){
    // input stationary
    int BLOCKS_PER_PIXEL = ceil(float(oc)/float(THREADS_PER_BLOCK));
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int cid = bid % BLOCKS_PER_PIXEL; // channel block index (within pixel)
    int pid = bid / BLOCKS_PER_PIXEL; // pixel index
    // compute output pixel of the block
    int h = pid % oh;
    int w = pid / oh;
    // declare on-chip shared memory
    extern __shared__ float M[];
    // read input data once per block (shared across threads)
    // this process could serve as bottleneck, load distribution is critical
    // distribute indices across threads
    int load_per_thread = ceil(float(kw*kh*ic)/float(THREADS_PER_BLOCK));
    int l = load_per_thread * tid;
    int u = load_per_thread * (tid + 1);
    if (l < kw*kh*ic) {
        u = (u < kw*kh*ic)? u : kw*kh*ic;
        for (int idx=l; idx<u; idx++){
            int k = idx%ic;
            int j = idx/ic%kh;
            int i = idx/ic/kh;
            M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] = I[INDEX_ROW_MAJOR_3(w*sw+i,h*sh+j,k, iw,ih,ic)];
        }
    }
    /*
    if(tid == 0){
        for (int i=0; i<kw; i++){
            for (int j=0; j<kh; j++){
                for (int k=0; k<ic; k++){
                    M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] = I[INDEX_ROW_MAJOR_3(w*sw+i,h*sh+j,k, iw,ih,ic)];
                }
            }
        }
    }
    */
    // compute block index in output channel dimension
    int ofs = cid * THREADS_PER_BLOCK;
    // handle boundary
    if (tid >= ((oc - ofs < THREADS_PER_BLOCK)? (oc - ofs) : THREADS_PER_BLOCK)) return;
    // wait until data is ready
    __syncthreads();
    // apply convolution
    float *o = R + INDEX_ROW_MAJOR_3(w,h,ofs+tid, ow,oh,oc);
    for (int i=0; i<kw; i++){
        for (int j=0; j<kh; j++){
            for (int k=0; k<ic; k++){
                atomicAdd(o, M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] * K[INDEX_ROW_MAJOR_4(i,j,k,ofs+tid, kw,kh,ic,oc)]);
            }
        }
    }
}
__global__ void conv_ws(float *I, float *K, float *R, int iw, int ih, int ow, int oh, int kw, int kh, int sw, int sh, int ic, int oc){
    // weight stationary
    int BLOCKS_PER_CHANNEL = ceil(float(ow * oh)/float(THREADS_PER_BLOCK));
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int pid = bid % BLOCKS_PER_CHANNEL; // pixel block index (within channel)
    int cid = bid / BLOCKS_PER_CHANNEL; // output channel index
    // declare on-chip shared memory
    extern __shared__ float M[];
    // read input data once per block (shared across threads)
    // this process could serve as bottleneck, load distribution is critical
    // distribute indices across threads
    int load_per_thread = ceil(float(kw*kh*ic)/float(THREADS_PER_BLOCK));
    int l = load_per_thread * tid;
    int u = load_per_thread * (tid + 1);
    if (l < kw*kh*ic) {
        u = (u < kw*kh*ic)? u : kw*kh*ic;
        for (int idx=l; idx<u; idx++){
            int k = idx%ic;
            int j = idx/ic%kh;
            int i = idx/ic/kh;
            M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] = K[INDEX_ROW_MAJOR_4(i,j,k,cid, kw,kh,ic,oc)];
        }
    }
    /*
    if(tid == 0){
        for (int i=0; i<kw; i++){
            for (int j=0; j<kh; j++){
                for (int k=0; k<ic; k++){
                    M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)] = K[INDEX_ROW_MAJOR_4(i,j,k,cid, kw,kh,ic,oc)];
                }
            }
        }
    }
    */
    // compute block index in output pixel dimension
    int ofs = pid * THREADS_PER_BLOCK;
    // handle boundary
    if (tid >= ((ow * oh - ofs < THREADS_PER_BLOCK)? (ow * oh - ofs) : THREADS_PER_BLOCK)) return;
    // retrieve output pixel
    int w = (ofs + tid)/oh;
    int h = (ofs + tid)%oh;
    float *o = R + INDEX_ROW_MAJOR_3(w,h,cid, ow,oh,oc);
    // wait until data is ready
    __syncthreads();
    // apply convolution
    for (int i=0; i<kw; i++){
        for (int j=0; j<kh; j++){
            for (int k=0; k<ic; k++){
                atomicAdd(o, I[INDEX_ROW_MAJOR_3(w*sw+i,h*sh+j,k, iw,ih,ic)] * M[INDEX_ROW_MAJOR_3(i,j,k, kw,kh,ic)]);
            }
        }
    }
}
extern "C"
void conv2d(float * I, float * K, float * R, int iw, int ih, int ow, int oh, int kw, int kh, int sw, int sh, int ic, int oc) {
    float *dev_I, *dev_K, *dev_R;
    // I: (iw * ih * ic), row major ordered
    // K: (kw * kh * ic * oc), row major ordered
    // R: (ow * oh * oc), row major ordered
    // todo: 2d convolution between I and K
    // loop over outer dimensions, and compute dot product in chunks of size 512
    // kernel function: convolution for a single sliding window
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_I, iw * ih * ic * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_K, kw * kh * ic * oc * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_R, ow * oh * oc * sizeof(float) ) );
    // copy the arrays to the GPU
    HANDLE_ERROR( hipMemcpy( dev_I, I, iw * ih * ic * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_K, K, kw * kh * ic * oc * sizeof(float), hipMemcpyHostToDevice ) );
    // how to organize blocks?
    // maximizing data reuse and parallelism within a block
    // dynamic on-chip memory allocation
    int BLOCK_MEMSIZE = kw * kh * ic * sizeof(float);
    if (ow*oh > THREADS_PER_BLOCK){
        // weight stationary
        // within a block, hold kernel and thread over output pixels
        int BLOCKS_PER_CHANNEL = ceil(float(ow * oh)/float(THREADS_PER_BLOCK));
        int BLOCKS = oc * BLOCKS_PER_CHANNEL;
        conv_ws<<<BLOCKS,THREADS_PER_BLOCK,BLOCK_MEMSIZE>>>(dev_I, dev_K, dev_R, iw, ih, ow, oh, kw, kh, sw, sh, ic, oc);
    }else{
        // input stationary
        // within a block, hold input and thread over output channels
        int BLOCKS_PER_PIXEL = ceil(float(oc)/float(THREADS_PER_BLOCK));
        conv_is<<<ow*oh*BLOCKS_PER_PIXEL,THREADS_PER_BLOCK,BLOCK_MEMSIZE>>>(dev_I, dev_K, dev_R, iw, ih, ow, oh, kw, kh, sw, sh, ic, oc);
    }
    // copy the array back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( R, dev_R, ow * oh * oc * sizeof(float), hipMemcpyDeviceToHost ) );
    // cleanup
    hipFree(dev_I); hipFree(dev_K); hipFree(dev_R);
}





__global__ void badd(float *I, float *B, float *R, int ow, int oh, int oc){
    int BLOCKS_PER_CHANNEL = ceil(float(ow * oh)/float(THREADS_PER_BLOCK));
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int pid = bid % BLOCKS_PER_CHANNEL; // pixel block index (within channel)
    int cid = bid / BLOCKS_PER_CHANNEL; // channel index
    // compute block index in output pixel dimension
    int ofs = pid * THREADS_PER_BLOCK;
    // handle boundary
    if (tid >= ((ow * oh - ofs < THREADS_PER_BLOCK)? (ow * oh - ofs) : THREADS_PER_BLOCK)) return;
    // retrieve output pixel
    int w = (ofs + tid)/oh;
    int h = (ofs + tid)%oh;
    // add
    ofs = INDEX_ROW_MAJOR_3(w,h,cid, ow,oh,oc);
    atomicAdd(R + ofs, I[ofs] + B[cid]);
}
extern "C"
void bias_add(float * I, float * B, float * R, int ow, int oh, int oc) {
    float *dev_I, *dev_B, *dev_R;
    // I: (ow * oh * oc), row major ordered
    // B: (oc)
    // R: (ow * oh * oc), row major ordered
    // todo: element-wise addition
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_I, ow * oh * oc * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_B, oc * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_R, ow * oh * oc * sizeof(float) ) );
    // copy the arrays to the GPU
    HANDLE_ERROR( hipMemcpy( dev_I, I, ow * oh * oc * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_B, B, oc * sizeof(float), hipMemcpyHostToDevice ) );
    // block = channel, thread over pixels
    int BLOCKS_PER_CHANNEL = ceil(float(ow*oh)/float(THREADS_PER_BLOCK));
    badd<<<oc*BLOCKS_PER_CHANNEL,THREADS_PER_BLOCK>>>(dev_I, dev_B, dev_R, ow, oh, oc);
    // copy the array back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( R, dev_R, ow * oh * oc * sizeof(float), hipMemcpyDeviceToHost ) );
    // cleanup
    hipFree(dev_I); hipFree(dev_B); hipFree(dev_R);
}






__global__ void lr(float *I, float *R, int ow, int oh, int oc){
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    // handle boundary
    int ofs = ow*oh*oc - bid*THREADS_PER_BLOCK;
    if (tid >= (ofs < THREADS_PER_BLOCK? ofs : THREADS_PER_BLOCK)) return;
    // add
    ofs = bid*THREADS_PER_BLOCK+tid;
    atomicAdd(R + ofs, I[ofs] * (I[ofs]>0? 1 : 0.1));
}
extern "C"
void leaky_relu(float * I, float * R, int ow, int oh, int oc) {
    float *dev_I, *dev_R;
    // I: (ow * oh * oc), row major ordered
    // R: (ow * oh * oc), row major ordered
    // todo: element-wise rectification
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_I, ow * oh * oc * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_R, ow * oh * oc * sizeof(float) ) );
    // copy the arrays to the GPU
    HANDLE_ERROR( hipMemcpy( dev_I, I, ow * oh * oc * sizeof(float), hipMemcpyHostToDevice ) );
    // block = channel, thread over pixels
    int BLOCKS = ceil(float(ow*oh*oc)/float(THREADS_PER_BLOCK));
    lr<<<BLOCKS,THREADS_PER_BLOCK>>>(dev_I, dev_R, ow, oh, oc);
    // copy the array back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( R, dev_R, ow * oh * oc * sizeof(float), hipMemcpyDeviceToHost ) );
    // cleanup
    hipFree(dev_I); hipFree(dev_R);
}





__global__ void bn(float *I, float *M, float *G, float *V, float *R, float eps, int ow, int oh, int oc){
    int BLOCKS_PER_CHANNEL = ceil(float(ow * oh)/float(THREADS_PER_BLOCK));
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int pid = bid % BLOCKS_PER_CHANNEL; // pixel block index (within channel)
    int cid = bid / BLOCKS_PER_CHANNEL; // channel index
    // compute block index in output pixel dimension
    int ofs = pid * THREADS_PER_BLOCK;
    // handle boundary
    if (tid >= ((ow * oh - ofs < THREADS_PER_BLOCK)? (ow * oh - ofs) : THREADS_PER_BLOCK)) return;
    // retrieve output pixel
    int w = (ofs + tid)/oh;
    int h = (ofs + tid)%oh;
    ofs = INDEX_ROW_MAJOR_3(w,h,cid, ow,oh,oc);
    // normalize
    atomicAdd(R + ofs, G[cid] * (I[ofs] - M[cid]) / (sqrt(V[cid]]) + eps));
}
extern "C"
void batch_norm(float * I, float * M, float * G, float * V, float * R, float eps, int ow, int oh, int oc){
    float *dev_I, *dev_M, *dev_G, *dev_V, *dev_R;
    // I: (ow * oh * oc), row major ordered
    // M, G, V, R: (oc)
    // R: (ow * oh * oc), row major ordered
    // todo: element-wise normalization
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_I, ow * oh * oc * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_M, oc * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_G, oc * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_V, oc * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_R, ow * oh * oc * sizeof(float) ) );
    // copy the arrays to the GPU
    HANDLE_ERROR( hipMemcpy( dev_I, I, ow * oh * oc * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_M, M, oc * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_G, G, oc * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_V, V, oc * sizeof(float), hipMemcpyHostToDevice ) );
    // block = channel, thread over pixels
    int BLOCKS_PER_CHANNEL = ceil(float(ow*oh)/float(THREADS_PER_BLOCK));
    bn<<<oc*BLOCKS_PER_CHANNEL,THREADS_PER_BLOCK>>>(dev_I, dev_M, dev_G, dev_V, dev_R, eps, ow, oh, oc);
    // copy the array back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( R, dev_R, ow * oh * oc * sizeof(float), hipMemcpyDeviceToHost ) );
    // cleanup
    hipFree(dev_I); hipFree(dev_M); hipFree(dev_G); hipFree(dev_V); hipFree(dev_R);
}





__global__ void mp(float *I, float *R, int iw, int ih, int kw, int kh, int sw, int sh, int ow, int oh, int oc){
    // input stationary
    int BLOCKS_PER_CHANNEL = ceil(float(ow * oh)/float(THREADS_PER_BLOCK));
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int pid = bid % BLOCKS_PER_CHANNEL; // pixel block index (within channel)
    int cid = bid / BLOCKS_PER_CHANNEL; // output channel index
    // compute block index in output pixel dimension
    int ofs = pid * THREADS_PER_BLOCK;
    // handle boundary
    if (tid >= ((ow * oh - ofs < THREADS_PER_BLOCK)? (ow * oh - ofs) : THREADS_PER_BLOCK)) return;
    // retrieve output pixel
    int w = (ofs + tid)/oh;
    int h = (ofs + tid)%oh;
    // apply pooling
    float v = -1e20;
    for (int i=0; i<kw; i++){
        for (int j=0; j<kh; j++){
            if (w*sw+i>=iw || h*sh+j>=ih) continue;
            int idx = INDEX_ROW_MAJOR_3(w*sw+i,h*sh+j,cid, iw,ih,oc);
            v = ((I[idx] > v)? I[idx] : v);
        }
    }
    atomicAdd(R + INDEX_ROW_MAJOR_3(w,h,cid, ow,oh,oc), v);
}
extern "C"
void max_pool(float * I, float * R, int iw, int ih, int kw, int kh, int sw, int sh, int ow, int oh, int oc) {
    float *dev_I, *dev_R;
    // I: (iw * ih * oc), row major ordered
    // R: (ow * oh * oc), row major ordered
    // todo: max-pooling
    // kernel function: pooling for a single sliding window
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_I, iw * ih * oc * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_R, ow * oh * oc * sizeof(float) ) );
    // copy the arrays to the GPU
    HANDLE_ERROR( hipMemcpy( dev_I, I, iw * ih * oc * sizeof(float), hipMemcpyHostToDevice ) );
    // within a block, thread over output pixels
    int BLOCKS_PER_CHANNEL = ceil(float(ow * oh)/float(THREADS_PER_BLOCK));
    mp<<<oc*BLOCKS_PER_CHANNEL,THREADS_PER_BLOCK>>>(dev_I, dev_R, iw, ih, kw, kh, sw, sh, ow, oh, oc);
    // copy the array back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( R, dev_R, ow * oh * oc * sizeof(float), hipMemcpyDeviceToHost ) );
    // cleanup
    hipFree(dev_I); hipFree(dev_R);
}