#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

extern "C"
void ki_apply(float *K, float *I, float *res, int in_size, int out_size) {
    // K: (in_size, out_size)
    // I: (1, in_size)
    // res: (1, out_size)
    printf("matmul: Started offloaded matmul in GPU... ");

    hipError_t cudaStat;     // cudaMalloc status
    hipblasStatus_t stat;      // CUBLAS functions status
    hipblasHandle_t handle;    // CUBLAS context

    // on the device
    float * d_I; 
    float * d_K;
    float * d_res;

    cudaStat = hipMalloc((void**)&d_I, 1*in_size*sizeof(*I));
    cudaStat = hipMalloc((void**)&d_K, in_size*out_size*sizeof(*K));
    cudaStat = hipMalloc((void**)&d_res, 1*out_size*sizeof(*res));
    stat = hipblasCreate(&handle);

    float al = 1.0f;
    float bet = 1.0f;

    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, out_size, in_size, &al, d_I, 1, d_K, in_size, &bet, d_res, 1);
    stat = hipblasGetMatrix(1, out_size, sizeof(*res), d_res, 1, res, 1); // cp d_c - > c

    // free device memory
    hipFree(d_I);
    hipFree(d_K);
    hipFree(d_res);
    // destroy CUBLAS context
    hipblasDestroy(handle);
    printf("finished");
}