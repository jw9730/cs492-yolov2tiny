#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

extern "C"
void ki_apply(float *K, float *I, float *R, int in_size, int out_size) {
    // K: (in_size, out_size)
    // I: (1, in_size)
    // R: (1, out_size)

    hipError_t cudaStat;     // cudaMalloc status
    hipblasStatus_t stat;      // CUBLAS functions status
    hipblasHandle_t handle;    // CUBLAS context

    // on the device
    float * d_I; 
    float * d_K;
    float * d_R;

    cudaStat = hipMalloc((void**)&d_I, 1 * in_size * sizeof(float));
    cudaStat = hipMalloc((void**)&d_K, in_size * out_size * sizeof(float));
    cudaStat = hipMalloc((void**)&d_R, 1 * out_size * sizeof(float));
    stat = hipblasCreate(&handle);

    float a = 1.0f;
    float b = 1.0f;

    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                       1, out_size, in_size,
                       &a, d_I, 1, d_K, in_size,
                       &b, d_R, 1);
    stat = hipblasGetMatrix(1, out_size, sizeof(float), d_R, 1, R, 1); // cp d_c - > c

    // free device memory
    hipFree(d_I);
    hipFree(d_K);
    hipFree(d_R);
    // destroy CUBLAS context
    hipblasDestroy(handle);
}