#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

extern "C"
void ki_apply(float *K, float *I, float *res, int in_size, int out_size) {
    // K: (in_size, out_size)
    // I: (1, in_size)
    // res: (1, out_size)

    hipError_t cudaStat;     // cudaMalloc status
    hipblasStatus_t stat;      // CUBLAS functions status
    hipblasHandle_t handle;    // CUBLAS context

    // on the device
    float * d_I; 
    float * d_K;
    float * d_res;

    cudaStat = hipMalloc(( void**)& d_I , 1*in_size*sizeof(*I));
    assert (cudaStat == 0);
    cudaStat = hipMalloc(( void**)& d_K, in_size*out_size*sizeof(*K));
    assert (cudaStat == 0);
    cudaStat = hipMalloc(( void**)& d_res , 1*out_size*sizeof(*res));
    assert (cudaStat == 0);
    stat = hipblasCreate(&handle);
    assert (stat == 0);

    float al =1.0f;
    float bet =0.0f;

    stat = hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,1,out_size,in_size,&al,d_I,1,d_K,in_size,&bet,d_res,1);
    assert (stat == 0);
    stat = hipblasGetMatrix(1 ,out_size , sizeof(*res) , d_res ,1 ,res , 1 ); // cp d_c - > c
    assert (stat == 0);

    hipFree(d_I);
    // free device memory
    hipFree(d_K);
    // free device memory
    hipFree(d_res);
    // free device memory
    hipblasDestroy( handle );
    // destroy CUBLAS context
}